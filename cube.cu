
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>

__global__ void cube(int *d_in, int *d_out) {
  int idx = threadIdx.x;
  d_out[idx] = d_in[idx] * d_in[idx] * d_in[idx];
}

int main() {
  const int ARRAY_SIZE = 128;

  // Allocate memory for arrays on host
  int *h_arr = new int[ARRAY_SIZE];

  // Initialize array on host
  for (int i = 0; i < ARRAY_SIZE; i++) {
    h_arr[i] = i;
  }

  // Allocate memory for array on device
  int *d_arr, *d_res;
  hipMalloc(&d_arr, ARRAY_SIZE * sizeof(int));
  hipMalloc(&d_res, ARRAY_SIZE * sizeof(int));

  // Copy array from host to device
  hipMemcpy(d_arr, h_arr, ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice);

  // Launch the kernel
  cube<<<1, ARRAY_SIZE>>>(d_arr, d_res);

  // Copy the result back to host
  hipMemcpy(h_arr, d_res, ARRAY_SIZE * sizeof(int), hipMemcpyDeviceToHost);

  // Print the result
  for (int i = 0; i < ARRAY_SIZE; i++) {
    std::cout << h_arr[i] << " ";
  }

  // Free the array on host and device
  hipFree(d_arr);
  hipFree(d_res);
  delete[] h_arr;

  return 0;
}